#include "hip/hip_runtime.h"
#include <iostream>
#include <type_traits>
#include <hip/hip_runtime.h>
#include <cuda/std/span>
#include <fastq/fastq.hpp>
#include <fastq/reader.hpp>
#include <fastq/splitter.hpp>
#include <fastq/barcode.hpp>


namespace cu {

  template <typename T>
  struct chunk_allocator {
    static_assert(std::is_trivially_constructible_v<T>);
    static_assert(std::is_trivially_destructible_v<T>);

    static void* alloc(size_t bytes) { 
      void* ptr; 
      if (hipSuccess != hipMallocManaged(&ptr, bytes)) {
        throw std::bad_alloc{};
      }
      std::cout << "0x" << std::hex << uintptr_t(ptr) << std::dec << " + " << bytes << std::endl;
      hipDeviceSynchronize();
      // hipMemAdvise(ptr, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
      // hipMemAdvise(ptr, bytes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
      return ptr;
    };

    static void free(void* ptr) noexcept { 
      std::cout << "0x" << std::hex << uintptr_t(ptr) << std::dec << " - " << std::endl;
      hipDeviceSynchronize();
      hipFree(ptr); 
    } 
  };


  using reader_t = fastq::detail::reader_t<chunk_allocator<char>, 16 * 1024, 16 * 1024 * 1024>;


  __host__ __device__
  struct str_view {
    const char* first;
    size_t length;
  };


  struct read_line_split {
    using value_type = str_view;

    static value_type apply(fastq::str_view& /* in out */ cv) noexcept {
      using field_split = fastq::policy::delim_split<char, '\n', 0, 1>;
      auto ret = value_type{};
      for (auto i = 0; i < 4; ++i) {
        auto field = field_split::apply(cv);
        if (i == 1) {
          ret = { field.begin(), field.length() };
        }
      }
      return ret;
    }
  };


  using read_field_splitter = fastq::base_splitter<
    cu::reader_t,
    fastq::chunk_splitter<
      fastq::policy::delim_chunk_trim<const char*, fastq::SeqDelim>,
      read_line_split
    >
  >;


  struct barcode_split {
    using value_type = str_view;

    static value_type apply(auto /* in out */ cv) noexcept {
      using line_split = fastq::policy::delim_split<char, '\n', 0, 1>;
      auto line = line_split::apply(cv);
      auto code = line.substr(line.find_last_of(" \t") + 1);
      return { code.begin(), code.length() };
    }
  };


  using barcode_splitter = fastq::base_splitter<
    reader_t,
    fastq::chunk_splitter<
      fastq::policy::delim_chunk_trim<char, '\n'>,
      barcode_split
    >
  >;


  template <typename T>
  auto make_device_ptr(const T* hptr, size_t n) {
    void* dptr = nullptr;
    if (auto res = hipMalloc(&dptr, n * sizeof(T)); hipSuccess != res) {
      throw std::bad_alloc();
    }
    hipMemcpy(dptr, hptr, n * sizeof(T), hipMemcpyHostToDevice);
    return std::shared_ptr<T[]>((T*)dptr, hipFree);
  }


  template <size_t N>
  __device__
  size_t edit_distance(const char* a, size_t m, const char* b, size_t n) {
    if (m > n) {
      auto s = b; b = a; a = s; 
      auto l = n; n = m; m = l; 
    }
    // remove matching prefixes and suffixes
    while (m && (*a == *b)) { ++a; ++b; --m; --n; }
    while (m && (a[m-1] == b[n-1])) { --m; --n; }
    size_t D[N + 1];  // scratch
    for (size_t i = 0; i < m + 1; ++i) D[i] = i;
    for (auto i = 1; i <= n; ++i) {
      const auto bi = b[i - 1];
      auto tmp = D[0]; D[0] = i;
      for (auto j = 1; j <= m; ++j) {
        if (a[j - 1] != bi) {
          tmp = min(D[j], min(D[j - 1], tmp)) + 1;
        }
        auto t = tmp; tmp = D[0]; D[0] = t;
      }
    }
    return D[m];
  }


  __global__
  void blk_edit_distance(str_view* RX, size_t nrx, str_view* BC, size_t nbc, size_t* out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > nrx) return;
    str_view rx = RX[idx];
    for (int i = 0; i < nbc; ++i) {
      str_view bc = BC[i];
      out[idx] = edit_distance<8>(rx.first, 7, bc.first, bc.length);
    }
  }

  __global__
  void nop(str_view* RX, size_t nrx, str_view* BC, size_t nbc, size_t* out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx > nrx) return;
    out[idx] = idx;
  }

  void launch_blk_edit_distance(auto& blk, str_view* dbc, size_t nbc, size_t* ed_out) {
    auto drx = make_device_ptr<str_view>(blk.data(), blk.size());
    dim3 numBlocks(blk.size() / 256);
    blk_edit_distance<<<numBlocks, 256>>>(drx.get(), blk.size(), dbc, nbc, ed_out);
  }

}


int main() {
  constexpr size_t N = 10000;
  auto s = cu::read_field_splitter("../data/_gen_I2_001.fastq.gz");
  auto bc = cu::barcode_splitter("../data/BC_A.txt")(N);
  auto dbc = cu::make_device_ptr(bc.data(), bc.size());
  void* ed_out = nullptr;
  hipMalloc(&ed_out, N * sizeof(cu::str_view));
  size_t items = 0;
  while (!s.eof()) {
    auto blk = s(N);
    cu::launch_blk_edit_distance(blk, dbc.get(), bc.size(), (size_t*)ed_out);
    items += blk.size();
  }
  std::cout << items << " items read, " << s.reader().tot_bytes() / (1000 * 1000) << " MB" << std::endl;
  return 0;
}
