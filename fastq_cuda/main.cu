#include <type_traits>
#include <hip/hip_runtime.h>
#include <cuda/std/span>
#include <fastq/reader.hpp>
#include <fastq/splitter.hpp>


namespace cu {

  template <typename T>
  struct chunk_allocator {
    static_assert(std::is_trivially_constructible_v<T>);
    static_assert(std::is_trivially_destructible_v<T>);

    static void* alloc(size_t bytes) { 
      void* ptr; 
      if (hipSuccess != hipMallocManaged(&ptr, bytes)) {
        throw std::bad_alloc{};
      }
      hipMemAdvise(ptr, bytes, hipMemAdviseSetPreferredLocation, hipCpuDeviceId);
      hipMemAdvise(ptr, bytes, hipMemAdviseSetAccessedBy, hipCpuDeviceId);
      return ptr;
    };

    static void free(void* ptr) noexcept { hipFree(ptr); } 
  };


  using reader_t = fastq::detail::reader_t<chunk_allocator<char>, 64 * 1024 * 1024>;
  //using reader_t = fastq::reader_t;

}

using splitter_t = fastq::seq_splitter<cu::reader_t>;


int main() {
  auto s = splitter_t("../data/_gen_I2_001.fastq.gz");
  while (!s.eof()) {
    auto x = s();
  }
  return 0;
}
